
#include <hip/hip_runtime.h>
#pragma region License
/*
The MIT License

Copyright (c) 2009 Sky Morey

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#pragma endregion
/*
#include <cuda.h>;
#include "..\Core.h";
#include "..\..\..\FeaServer.Engine.Cpu\src\Time\Scheduler\SliceCollection.hpp"
using namespace Time::Scheduler;

__global__ void Schedule()
{
	Element e;
	e.ScheduleStyle = Time::ElementScheduleStyle::Multiple;

	SliceCollection s;
	s.Schedule(&e, 10);
	s.MoveNextSlice();
}

int main()
{
	//fallocHeapInitialize(nullptr, 0);

	cudaPrintfInit();
	Schedule<<<1, 1>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	printf("\ndone.\n"); scanf("%c");
    return 0;
}

#include "..\..\..\FeaServer.Engine.Cpu\src\Time\Scheduler\SliceCollection.hpp"
*/