#pragma region License
/*
The MIT License

Copyright (c) 2009 Sky Morey

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#pragma endregion
#ifndef CUFALLOC_C
#define CUFALLOC_C

#include "cuFalloc.cuh"
//#include <sm_11_atomic_functions.h>

// This is the smallest amount of memory, per-thread, which is allowed.
// It is also the largest amount of space a single printf() can take up
const static int HEAPCHUNK_SIZE = 128;
const static int FALLOCNODE_SLACK = 0x10;

// This is the header preceeding all printf entries.
// NOTE: It *must* be size-aligned to the maximum entity size (size_t)
typedef struct __align__(8) _cuFallocHeapChunk {
    unsigned short magic;				// Magic number says we're valid
    volatile struct _cuFallocHeapChunk* next;	// Next chunk pointer
} cuFallocHeapChunk;

typedef struct __align__(8) _cuFallocDeviceHeap {
	unsigned short chunks;
	volatile cuFallocHeapChunk* freeChunks;
} fallocDeviceHeap;

typedef struct _cuFallocDeviceNode {
	struct _cuFallocDeviceNode* next;
	struct _cuFallocDeviceNode* nextAvailable;
	unsigned short freeOffset;
	unsigned short magic;
} cuFallocDeviceNode;

typedef struct _cuFallocContext {
	cuFallocDeviceNode node;
	cuFallocDeviceNode* nodes;
	cuFallocDeviceNode* availableNodes;
	fallocDeviceHeap* deviceHeap;
} fallocContext;

// All our headers are prefixed with a magic number so we know they're ready
#define CUFALLOC_MAGIC (unsigned short)0x3412        // Not a valid ascii character
#define CUFALLOCNODE_MAGIC (unsigned short)0x7856

__device__ void fallocInit(fallocDeviceHeap* deviceHeap) {
	if (threadIdx.x != 0)
		return;
	volatile cuFallocHeapChunk* chunk = (cuFallocHeapChunk*)((__int8*)deviceHeap + sizeof(fallocDeviceHeap));
	deviceHeap->freeChunks = chunk;
	unsigned short chunks = deviceHeap->chunks;
	// preset all chunks
	chunk->magic = CUFALLOC_MAGIC;
	while (chunks-- > 1)
	{
		chunk = chunk->next = (cuFallocHeapChunk*)((__int8*)chunk + sizeof(cuFallocHeapChunk) + HEAPCHUNK_SIZE);
		chunk->magic = CUFALLOC_MAGIC;
	}
	chunk->next = nullptr;
	chunk->magic = CUFALLOC_MAGIC;
}

__device__ void* fallocGetChunk(fallocDeviceHeap* deviceHeap) {
	if (threadIdx.x != 0)
		__THROW;
	volatile cuFallocHeapChunk* chunk = deviceHeap->freeChunks;
	if (chunk == nullptr)
		return nullptr;
	{ // critical
		deviceHeap->freeChunks = chunk->next;
		chunk->next = nullptr;
	}
	return (void*)((short*)chunk + sizeof(cuFallocHeapChunk));
}

__device__ void fallocFreeChunk(fallocDeviceHeap* deviceHeap, void* obj) {
	if (threadIdx.x != 0)
		__THROW;
	cuFallocHeapChunk* chunk = (cuFallocHeapChunk*)((__int8*)obj - sizeof(cuFallocHeapChunk));
	if (chunk->magic != CUFALLOC_MAGIC)
		__THROW;
	{ // critical
		chunk->next = deviceHeap->freeChunks;
		deviceHeap->freeChunks = chunk;
	}
}

//////////////////////
// ALLOC

__device__ static fallocContext* fallocCreateCtx(fallocDeviceHeap* deviceHeap) {
	if (sizeof(fallocContext) > HEAPCHUNK_SIZE)
		__THROW;
	fallocContext* ctx = (fallocContext*)fallocGetChunk(deviceHeap);
	if (ctx == nullptr)
		__THROW;
	ctx->deviceHeap = deviceHeap;
	unsigned short freeOffset = ctx->node.freeOffset = sizeof(fallocContext);
	ctx->node.magic = CUFALLOCNODE_MAGIC;
	ctx->node.next = nullptr; ctx->nodes = (cuFallocDeviceNode*)ctx;
	ctx->node.nextAvailable = nullptr; ctx->availableNodes = (cuFallocDeviceNode*)ctx;
	// close node
	if ((freeOffset + FALLOCNODE_SLACK) > HEAPCHUNK_SIZE)
		ctx->availableNodes = nullptr;
	return ctx;
}

__device__ static void fallocDisposeCtx(fallocContext* ctx) {
	fallocDeviceHeap* deviceHeap = ctx->deviceHeap;
	for (cuFallocDeviceNode* node = ctx->nodes; node != nullptr; node = node->next)
		fallocFreeChunk(deviceHeap, node);
}

__device__ static void* falloc(fallocContext* ctx, unsigned short bytes, bool alloc) {
	if (bytes > (HEAPCHUNK_SIZE - sizeof(fallocContext)))
		__THROW;
	// find or add available node
	cuFallocDeviceNode* node;
	unsigned short freeOffset;
	unsigned char hasFreeSpace;
	cuFallocDeviceNode* lastNode;
	for (lastNode = (cuFallocDeviceNode*)ctx, node = ctx->availableNodes; node != nullptr; lastNode = node, node = (alloc ? node->nextAvailable : node->next))
		 if (hasFreeSpace = ((freeOffset = (node->freeOffset + bytes)) <= HEAPCHUNK_SIZE))
			 break;
	if ((node == nullptr) || !hasFreeSpace) {
		// add node
		node = (cuFallocDeviceNode*)fallocGetChunk(ctx->deviceHeap);
		if (node == nullptr)
			__THROW;
		freeOffset = node->freeOffset = sizeof(cuFallocDeviceNode); 
		freeOffset += bytes;
		node->magic = CUFALLOCNODE_MAGIC;
		node->next = ctx->nodes; ctx->nodes = node;
		node->nextAvailable = (alloc ? ctx->availableNodes : nullptr); ctx->availableNodes = node;
	}
	//
	void* obj = (__int8*)node + node->freeOffset;
	node->freeOffset = freeOffset;
	// close node
	if (alloc && ((freeOffset + FALLOCNODE_SLACK) > HEAPCHUNK_SIZE)) {
		if (lastNode == (cuFallocDeviceNode*)ctx)
			ctx->availableNodes = node->nextAvailable;
		else
			lastNode->nextAvailable = node->nextAvailable;
		node->nextAvailable = nullptr;
	}
	return obj;
}

__device__ static void* fallocRetract(fallocContext* ctx, unsigned short bytes) {
	cuFallocDeviceNode* node = ctx->availableNodes;
	int freeOffset = (int)node->freeOffset - bytes;
	// multi node, retract node
	if ((node != &ctx->node) && (freeOffset < sizeof(cuFallocDeviceNode)))
	{
		node->freeOffset = sizeof(cuFallocDeviceNode);
		// search for previous node
		cuFallocDeviceNode* lastNode;
		for (lastNode = (cuFallocDeviceNode*)ctx, node = ctx->nodes; node != nullptr; lastNode = node, node = node->next)
			if (node == ctx->availableNodes)
				break;
		node = ctx->availableNodes = lastNode;
		freeOffset = (int)node->freeOffset - bytes;
	}
	// first node && !overflow
	if ((node == &ctx->node) && (freeOffset < sizeof(fallocContext)))
		__THROW;
	node->freeOffset = (unsigned short)freeOffset;
	return (__int8*)node + freeOffset;
}

__device__ static void fallocMark(fallocContext* ctx, void* &mark, unsigned short &mark2) { mark = ctx->availableNodes; mark2 = ctx->availableNodes->freeOffset; }
__device__ static bool fallocAtMark(fallocContext* ctx, void* mark, unsigned short mark2) { return ((mark == ctx->availableNodes) && (mark2 == ctx->availableNodes->freeOffset)); }


///////////////////////////////////////////////////////////////////////////////
// HOST SIDE

//
//  cudaFallocInit
//
//  Takes a buffer length to allocate, creates the memory on the device and
//  returns a pointer to it for when a kernel is called. It's up to the caller
//  to free it.
//
extern "C" cudaFallocHeap cudaFallocInit(size_t bufferLen, hipError_t* error) {
	cudaFallocHeap heap; memset(&heap, 0, sizeof(cudaFallocHeap));
	// Fix up chunkSize to include cpuFallocHeapChunk
	int chunkSize = sizeof(cuFallocHeapChunk) + HEAPCHUNK_SIZE;
	if ((chunkSize % 16) > 0)
        chunkSize += (16 - (chunkSize % 16));
    // Fix up bufferlen to be a multiple of chunkSize
    bufferLen = (bufferLen < chunkSize ? chunkSize : bufferLen);
    if ((bufferLen % chunkSize) > 0)
        bufferLen += (chunkSize - (bufferLen % chunkSize));
	unsigned short chunks = bufferLen / chunkSize;
	// Fix up bufferlen to include cudaFallocHeap
	bufferLen += sizeof(cudaFallocHeap);
	if ((bufferLen % 16) > 0)
        bufferLen += (16 - (bufferLen % 16));
    // Allocate a print buffer on the device and zero it
	fallocDeviceHeap* deviceHeap;
	if ( ((error == nullptr) && (hipMalloc((void**)&deviceHeap, bufferLen) != hipSuccess)) ||
		((error != nullptr) && ((*error = hipMalloc((void**)&deviceHeap, bufferLen)) != hipSuccess)) )
		return heap;
    hipMemset(deviceHeap, 0, bufferLen);
	// transfer to deviceHeap
	fallocDeviceHeap hostHeap;
	hostHeap.freeChunks = nullptr;
	hostHeap.chunks = chunks;
	hipMemcpy(deviceHeap, &hostHeap, sizeof(fallocDeviceHeap), hipMemcpyHostToDevice);
	// return deviceHeap
	if (error != nullptr)
		*error = hipSuccess;
	heap.deviceHeap = deviceHeap;
	heap.length = (int)bufferLen;
    return heap;
}

//
//  cudaFallocEnd
//
//  Frees up the memory which we allocated
//
extern "C" void cudaFallocEnd(cudaFallocHeap &heap) {
    if (!heap.deviceHeap)
        return;
    hipFree(heap.deviceHeap); heap.deviceHeap = nullptr;
}


#endif // CUFALLOC_C